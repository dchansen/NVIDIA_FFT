#include <iostream>
#include <hipfft/hipfft.h>
#include <vector>
#include <chrono>

#include <fftw3.h>
#include <thread>

constexpr int NX = 256;
constexpr int NY = 256;
constexpr int RANK = 2;
constexpr int BATCHES = 16*32;
constexpr int SLICE_SIZE = NX * NY;
constexpr size_t BYTES = size_t(NX) * size_t(NY) * size_t(BATCHES)*sizeof(hipfftComplex);

void runFFT(hipfftComplex *input, hipfftComplex *output)
{

    std::vector<int> n = {NX, NY};
    hipfftHandle plan;
    if (hipfftPlanMany(&plan, RANK, n.data(), nullptr, 1, SLICE_SIZE, nullptr, 1, SLICE_SIZE, hipfftType::HIPFFT_C2C, BATCHES) != HIPFFT_SUCCESS)
    {
        //if (hipfftPlanMany(&plan, RANK, n.data(), n.data(), 1,SLICE_SIZE,n.data(),1,SLICE_SIZE,hipfftType::HIPFFT_C2C,BATCHES) != HIPFFT_SUCCESS){
        std::cout << "CUFFT Plan many failed " << std::endl;
    }

    hipfftExecC2C(plan, input, output, HIPFFT_FORWARD);
    hipDeviceSynchronize();

    if (hipfftDestroy(plan) != HIPFFT_SUCCESS)
    {
        std::cout << "Faield destroying plan" << std::endl;
    }
}

void runFFTW(fftwf_complex *in, fftwf_complex *out)
{

    std::vector<int> n = {NX, NY};
    auto plan = fftwf_plan_many_dft(2, n.data(),BATCHES, in,nullptr, 1,SLICE_SIZE, out, nullptr,1,SLICE_SIZE,1, FFTW_ESTIMATE);

    fftwf_execute_dft(plan, in, out);
    fftwf_destroy_plan(plan);
}

int main(int, char **)
{
    std::cout << "Hello, world!\n";


    {
        hipfftComplex *input;
        hipfftComplex *output;
        hipMalloc(&input, BYTES);
        hipMalloc(&output, BYTES);

        auto start = std::chrono::high_resolution_clock::now();

        constexpr int repetitions = 10;

        for (int i = 0; i < repetitions; i++)
        {
            runFFT(input, output);
        }

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end - start;
        std::cout << "Time per FFT " << elapsed_seconds.count() / repetitions << "s" << std::endl;

        hipFree(input);
        hipFree(output);
    }

    {
        fftwf_complex* input = (fftwf_complex*)malloc(BYTES);
        fftwf_complex* output = (fftwf_complex*)malloc(BYTES);
        
        constexpr int repetitions = 10;
        fftwf_init_threads();
        fftwf_plan_with_nthreads(std::thread::hardware_concurrency());

        auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < repetitions; i++)
        {
            runFFTW(input, output);
        }

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end - start;
        std::cout << "Time per FFT " << elapsed_seconds.count() / repetitions << "s" << std::endl;
        free(input);
        free(output);

    }

}